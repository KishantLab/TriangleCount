#include <iostream>
#include <hip/hip_runtime.h>

const int N = 4;  // Number of vertices in the original graph

// Helper function to print CSR format
void printCSR(int* values, int* row_ptr, int* col_indices, int num_rows) {
    std::cout << "Values: ";
    for (int i = 0; i < row_ptr[num_rows]; ++i) {
        std::cout << values[i] << " ";
    }
    std::cout << std::endl;

    std::cout << "Row Pointer: ";
    for (int i = 0; i <= num_rows; ++i) {
        std::cout << row_ptr[i] << " ";
    }
    std::cout << std::endl;

    std::cout << "Column Indices: ";
    for (int i = 0; i < row_ptr[num_rows]; ++i) {
        std::cout << col_indices[i] << " ";
    }
    std::cout << std::endl;
}

// CUDA kernel to construct induced subgraph using CSR format
__global__ void constructSubgraphCSR(int* originalValues, int* originalRowPtr, int* originalColIndices,
                                      int* subgraphValues, int* subgraphRowPtr, int* subgraphColIndices,
                                      int* vertices, int numVertices) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;

    if (tid < numVertices) {
        int subgraphIdx = vertices[tid];
        subgraphRowPtr[tid + 1] = originalRowPtr[subgraphIdx + 1] - originalRowPtr[subgraphIdx];

        for (int i = originalRowPtr[subgraphIdx]; i < originalRowPtr[subgraphIdx + 1]; ++i) {
            int originalIdx = originalColIndices[i];
            subgraphColIndices[i - originalRowPtr[subgraphIdx]] = originalIdx;
            subgraphValues[i - originalRowPtr[subgraphIdx]] = originalValues[i];
        }
    }
}

int main() {
    // Define the original graph in CSR format
    int originalValues[] = {1, 1, 1, 1, 1, 1, 1};
    int originalRowPtr[] = {0, 2, 4, 6, 7};
    int originalColIndices[] = {1, 2, 0, 2, 0, 1, 3};

    // Define the vertices of the induced subgraph
    int subgraphVertices[] = {1, 2, 3};  // Example: vertices 1, 2, and 3

    // Allocate device memory
    int* d_originalValues, * d_originalRowPtr, * d_originalColIndices;
    int* d_subgraphValues, * d_subgraphRowPtr, * d_subgraphColIndices;
    int* d_vertices;

    hipMalloc((void**)&d_originalValues, sizeof(originalValues));
    hipMalloc((void**)&d_originalRowPtr, sizeof(originalRowPtr));
    hipMalloc((void**)&d_originalColIndices, sizeof(originalColIndices));

    hipMalloc((void**)&d_subgraphValues, sizeof(originalValues));
    hipMalloc((void**)&d_subgraphRowPtr, sizeof(subgraphVertices) + 1);
    hipMalloc((void**)&d_subgraphColIndices, sizeof(originalColIndices));

    hipMalloc((void**)&d_vertices, sizeof(subgraphVertices));

    // Copy data to device
    hipMemcpy(d_originalValues, originalValues, sizeof(originalValues), hipMemcpyHostToDevice);
    hipMemcpy(d_originalRowPtr, originalRowPtr, sizeof(originalRowPtr), hipMemcpyHostToDevice);
    hipMemcpy(d_originalColIndices, originalColIndices, sizeof(originalColIndices), hipMemcpyHostToDevice);

    hipMemcpy(d_vertices, subgraphVertices, sizeof(subgraphVertices), hipMemcpyHostToDevice);

    // Launch the kernel
    int numBlocks = 1;
    int threadsPerBlock = N;
    constructSubgraphCSR<<<numBlocks, threadsPerBlock>>>(d_originalValues, d_originalRowPtr, d_originalColIndices,
                                                          d_subgraphValues, d_subgraphRowPtr, d_subgraphColIndices,
                                                          d_vertices, sizeof(subgraphVertices));

    // Copy result back to host
    int subgraphValues[N];
    int subgraphRowPtr[N + 1];
    int subgraphColIndices[N];

    hipMemcpy(subgraphValues, d_subgraphValues, sizeof(subgraphValues), hipMemcpyDeviceToHost);
    hipMemcpy(subgraphRowPtr, d_subgraphRowPtr, sizeof(subgraphRowPtr), hipMemcpyDeviceToHost);
    hipMemcpy(subgraphColIndices, d_subgraphColIndices, sizeof(subgraphColIndices), hipMemcpyDeviceToHost);

    // Print the induced subgraph in CSR format
    std::cout << "Original Graph (CSR format):" << std::endl;
    printCSR(originalValues, originalRowPtr, originalColIndices, N);

    std::cout << "\nInduced Subgraph (CSR format):" << std::endl;
    printCSR(subgraphValues, subgraphRowPtr, subgraphColIndices, sizeof(subgraphVertices));

    // Free device memory
    hipFree(d_originalValues);
    hipFree(d_originalRowPtr);
    hipFree(d_originalColIndices);
    hipFree(d_subgraphValues);
    hipFree(d_subgraphRowPtr);
    hipFree(d_subgraphColIndices);
    hipFree(d_vertices);

    return 0;
}

