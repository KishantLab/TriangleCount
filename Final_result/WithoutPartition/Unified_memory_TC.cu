#include "hip/hip_runtime.h"
#include<stdio.h>
#include<stdlib.h>
#include<hip/hip_runtime.h>
#include<math.h>
#include<time.h>
#include <hipcub/hipcub.hpp>

#define N_THREADS_PER_BLOCK 256
#define SHARED_MEM 256

//-------------------intersection function ----------------------------------
__device__ __forceinline__ int Search (int skey , int *neb, int sizelist)
{
	int total = 0;
	if(skey < neb[0] || skey > neb[sizelist])
	{
		return 0;
	}
	else if(skey == neb[0] || skey == neb[sizelist])
	{
		return 1;
	}
	else
	{
		int lo = 1;
		int hi = sizelist-1;
		int mid=0;
		while( lo <= hi)
		{
			mid = (hi+lo)/2;
			//printf("\nskey :%d , mid : %d ",skey,neb[mid]);
			if( neb[mid] < skey){lo=mid+1;}
			else if(neb[mid] > skey){hi=mid-1;}
			else if(neb[mid] == skey)
			{
				total++;
				break;
			}
		}
	}
	return total;
}
__global__ void Find_Triangle(int *g_col_index, int *g_row_ptr, int vertex, int edge ,unsigned long long int *g_sum )
{
	//int id = threadIdx.x + blockIdx.x * blockDim.x ; //Define id with thread id
	int bid = blockIdx.x;
	int tid = threadIdx.x;
	__shared__ int start;
	__shared__ int end;
	 int triangle;
	__shared__ int neb[SHARED_MEM];
	__shared__ unsigned long long int s_sum[N_THREADS_PER_BLOCK];
	//int start = g_row_ptr[bid];
	//int end = g_row_ptr[bid+1]-1;
	//int index = reordered_array[bid];
	if(tid ==0)
	{
		triangle = 0;
		start = g_row_ptr[bid];
		end = g_row_ptr[bid+1]-1;
	}
	__syncthreads();
	int size_list1 = end - start;
	if(size_list1 < 1)
	{
		g_sum[bid] = 0;
	}
	else
	{
	//if(size_list1 ==0 ) return;
			if(size_list1 < N_THREADS_PER_BLOCK)
			{
				if(tid <= size_list1)
				{
					neb[tid] = g_col_index[tid+start];
				}
				__syncthreads();
				for( int i = 0; i <= size_list1; i++)
				{
					int start2 = g_row_ptr[neb[i]];
					int end2 = g_row_ptr[neb[i]+1]-1;
					int size_list2 = end2 - start2;
					int M = ceil((float)(size_list2 +1)/N_THREADS_PER_BLOCK);
					#pragma unroll
					for( int k = 0; k < M; k++)
					{
						int id = N_THREADS_PER_BLOCK * k + tid;
						if(id <= size_list2)
						{
							int result = 0;
							result = Search(g_col_index[id+start2],neb,size_list1);
							//printf("\nedge(%d , %d) : %d , tid : %d, size_list1 :%d , size_list2: %d, start2 :%d , end2 :%d skey:%d, neb[0]:%d ,neb[%d]:%d",bid, neb[i], result,tid,size_list1+1,size_list2+1,start2,end2,g_col_index[id+start2],neb[0],size_list1,neb[size_list1]);
							//atomicAdd(&g_sum[0],result);
							//printf("\nedge(%d , %d) src : %d dst :%d ", bid,neb[i],size_list1+1,size_list2+1);
							triangle += result;
						}
					}
				}
			}
			else
			{
				int N = ceil((float)(size_list1 +1)/ N_THREADS_PER_BLOCK);
				int remining_size = size_list1;
				int size = N_THREADS_PER_BLOCK-1;
				for( int i = 0; i < N; i++)
				{
					int id = N_THREADS_PER_BLOCK * i + tid;
					if( remining_size > size)
					{
						if(id <= size_list1)
						{
							neb[tid] = g_col_index[id+start];
							//printf(" neb : %d", neb[tid]);
						}
						__syncthreads();
						for( int j = start; j <= end; j++)
						{
							int start2 = g_row_ptr[g_col_index[j]];
							int end2 = g_row_ptr[g_col_index[j]+1]-1;
							int size_list2 = end2 - start2;
							int M = ceil((float)(size_list2 +1)/N_THREADS_PER_BLOCK);
							#pragma unroll
							for( int k = 0; k < M; k++)
							{
								int tempid = N_THREADS_PER_BLOCK * k + tid;
								if(tempid <= size_list2)
								{
									int result = 0;
									result = Search(g_col_index[tempid+start2],neb,size);
									//printf("\nedge(%d , %d) : %d , tid : %d, size_list1 :%d , size_list2: %d, start2 :%d , end2 :%d, id :%d, skey :%d, N:%d, I:%d, remining_size:%d, size:%d, neb[0]:%d, neb[%d]:%d if ",bid, g_col_index[j], result,tid,size_list1+1,size_list2+1,start2,end2,id,g_col_index[tempid+start2],N,i,remining_size,size,neb[0],size,neb[size]);
									//atomicAdd(&g_sum[0],result);
									//printf("\nedge(%d , %d) src : %d dst :%d ", bid,g_col_index[j],size_list1+1,size_list2+1);
									triangle += result;
								}
							}
						}
						__syncthreads();
						remining_size = remining_size-(size+1);
					}
					else
					{

						if(id <= size_list1)
						{
							neb[tid] = g_col_index[id+start];
							//printf(" neb : %d", neb[tid]);
						}
						__syncthreads();
						for( int j = start; j <= end; j++)
						{
							int start2 = g_row_ptr[g_col_index[j]];
							int end2 = g_row_ptr[g_col_index[j]+1]-1;
							int size_list2 = end2 - start2;
							int M = ceil((float)(size_list2 +1)/ N_THREADS_PER_BLOCK);
							#pragma unroll
							for (int k = 0; k < M; k++)
							{
								int tempid = N_THREADS_PER_BLOCK * k + tid;
								if(tempid <= size_list2)
								{
									int result = 0;
									result = Search(g_col_index[tempid+start2],neb,remining_size);
									//printf("\nedge(%d , %d) : %d , tid : %d, size_list1 :%d , size_list2: %d, start2 :%d , end2 :%d, id :%d, skey :%d, N:%d, I:%d neb[0]:%d, neb[%d]:%d, else",bid, g_col_index[j], result,tid,size_list1+1,size_list2+1,start2,end2,id,g_col_index[tempid+start2],N,i,neb[0],remining_size,neb[remining_size]);
									//atomicAdd(&g_sum[0],result);
									//printf("\nedge(%d , %d) src : %d dst :%d ", bid,g_col_index[j],size_list1+1,size_list2+1);
									triangle += result;
								}
							}
						}
					}
					//__syncthreads();
				}
			}
	//	atomicAdd(&g_sum[0],triangle);
	 	s_sum[tid] = triangle;
    __syncthreads();
    if (tid == 0)
    {
        unsigned long long int block_sum = 0;
				#pragma unroll
        for (int i = 0; i < N_THREADS_PER_BLOCK; i++)
        {
            block_sum += s_sum[i];
        }
        g_sum[bid] = block_sum;
    }
	}
		// if(tid ==0)
		// g_sum[bid] = triangle;
	//	printf("%llu",triangle);
}
int main(int argc, char *argv[])
{
	int Edges=0,data=0,Vertex=0, row_ptr_s=0, col_idx_s=0; //vertex=10670, data allocation from file..
//**********file operations***************
	FILE *file;
	file = fopen(argv[1],"r");

	//******************Data From File*******************
	if(file == NULL)
	{
		printf("file not opened\n");
		exit(0);
	}
	else
	{
    fscanf(file , "%d", &Vertex);
    fscanf(file , "%d", &Edges);
		fscanf(file , "%d", &row_ptr_s);
		fscanf(file , "%d", &col_idx_s);

		//  int *row_ptr;  //CPU MEMORY ALLOCATION
		// row_ptr = ( int *)malloc(sizeof( int)*row_ptr_s);
   // int *col_index;   //CPU MEMORY ALLOCATION
   //  col_index = ( int *)malloc(sizeof( int)*col_idx_s);
   //
		// //printf("\nRow_ptr :");
		// for( int i=0; i<row_ptr_s; i++)
		// {
		// 	fscanf(file, "%d", &data);
		// 	row_ptr[i]=data;
		// 	//printf(" %llu",data);
		// }
		// //printf("\nCol_index :");
		// for( int j=0; j<col_idx_s; j++)
		// {
		// 	fscanf(file,"%d", &data);
		// 	col_index[j]=data;
		// 	//printf(" %llu",data);
		// }

		 int *g_row_ptr;   // GPU MEMORY ALLOCATION
		hipMallocManaged(&g_row_ptr,sizeof( int)*row_ptr_s);
     int *g_col_index;  //GPU MEMORY ALOOCATION
		hipMallocManaged(&g_col_index,sizeof( int)*col_idx_s);

		//**** Load DATA CPU TO GPU *********************
    // hipMemcpy(g_row_ptr,row_ptr,sizeof( int)*row_ptr_s,hipMemcpyHostToDevice);
		// hipMemcpy(g_col_index,col_index,sizeof( int)*col_idx_s,hipMemcpyHostToDevice);
    //printf("\nRow_ptr :");
    for( int i=0; i<row_ptr_s; i++)
    {
      fscanf(file, "%d", &data);
      g_row_ptr[i]=data;
      //printf(" %llu",data);
    }
    //printf("\nCol_index :");
    for( int j=0; j<col_idx_s; j++)
    {
      fscanf(file,"%d", &data);
      g_col_index[j]=data;
      //printf(" %llu",data);
    }
		//****************KERNEL CALLED *****************
		float total_exe_time = 0;
		for(int i=0; i < 3; i++)
		{

				hipEvent_t start3,stop3;
				hipEventCreate(&start3);
				hipEventCreate(&stop3);

				hipEvent_t start_sum,stop_sum;
				hipEventCreate(&start_sum);
				hipEventCreate(&stop_sum);

				unsigned long long int *g_sum;
				hipMallocManaged((void**)&g_sum,sizeof(unsigned long long int)*Vertex);

				unsigned long long int *out;
				out = (unsigned long long int *)malloc(sizeof(unsigned long long int)*1);

				unsigned long long int *d_out;
				hipMallocManaged((void**)&d_out,sizeof(unsigned long long int)*1);

				hipEventRecord(start3);
				Find_Triangle<<<Vertex,N_THREADS_PER_BLOCK>>>(g_col_index,g_row_ptr,Vertex,Edges,g_sum);
				hipEventRecord(stop3);
				hipDeviceSynchronize();
				hipEventSynchronize(stop3);
				//hipMemcpy(sum,g_sum,sizeof(unsigned long long int)*Vertex,hipMemcpyDeviceToHost);
				//unsigned long long int Triangle = sum[0];
				hipEventRecord(start_sum);
				void *d_temp_storage = NULL;
				size_t temp_storage_bytes = 0;
				hipcub::DeviceReduce::Sum(d_temp_storage, temp_storage_bytes, g_sum, d_out, Vertex);
				// Allocate temporary storage
				hipMallocManaged(&d_temp_storage, temp_storage_bytes);
				// Run sum-reduction
				hipcub::DeviceReduce::Sum(d_temp_storage, temp_storage_bytes, g_sum, d_out, Vertex);
				// d_out <-- [38]
				hipEventRecord(stop_sum);
				hipDeviceSynchronize();
				hipMemcpy(out,d_out,sizeof(unsigned long long int)*1,hipMemcpyDeviceToHost);
				unsigned long long int Triangle = out[0];
				hipEventSynchronize(stop_sum);
				float milliseconds = 0;
				hipEventElapsedTime(&milliseconds, start3, stop3);
				//printf("\nSearch : %.4f sec ",milliseconds/1000);
				float milliseconds_sum = 0;
				hipEventElapsedTime(&milliseconds_sum, start_sum, stop_sum);
				float total_time = (milliseconds/1000) + (milliseconds_sum/1000);
				printf("\nSearch : %.6f sec Vertex : %d Edge : %d Triangle : %llu  Sum_result : %.6f Sec, total_time : %.6f Sec\n",milliseconds/1000, Vertex, col_idx_s, Triangle, milliseconds_sum/1000, total_time);
				total_exe_time = total_exe_time + total_time;
				hipFree(g_sum);
				hipFree(d_out);
				free(out);
		}
		printf("\nTotal avg of 3 RUNS : %.6f Sec\n",total_exe_time/3);
		//********** FREE THE MEMORY BLOCKS *****************
		// free(col_index);
		// free(row_ptr);
		//free(sum);
		hipFree(g_col_index);
		hipFree(g_row_ptr);
		// hipFree(g_sum);
		// hipFree(d_out);
		// free(out);

	}
	//printf("\n");
	return 0;
}
