#include "hip/hip_runtime.h"
#include<stdio.h>
#include<stdlib.h>
#include<hip/hip_runtime.h>
#include<math.h>
#include<time.h>

#define NUM_VERTICES 999999999
#define NUM_EDGES 999999999
#define N_THREADS_PER_BLOCK 256
#define SHARED_MEM 256

//-------------------intersection function ----------------------------------
__device__ int Search (int skey , int *neb, int sizelist, int lo, int hi)
{
	int total = 0;
	if(skey < neb[0] || skey > neb[sizelist])
	{
		return 0;
	}
	else if(skey == neb[0] || skey == neb[sizelist])
	{
		return 1;
	}
	else
	{
		int lo = 1;
		int hi = sizelist-1;
		int mid=0;
		while( lo <= hi)
		{
			mid = (hi+lo)/2;
			//printf("\nskey :%d , mid : %d ",skey,neb[mid]);
			if( neb[mid] < skey){lo=mid+1;}
			else if(neb[mid] > skey){hi=mid-1;}
			else if(neb[mid] == skey)
			{
				total++;
				break;
			}
		}
	}
	return total;
}
__global__ void Find_Triangle(int *g_col_index, int *g_row_ptr, int vertex, int edge ,int *g_sum )
{
	//int id = threadIdx.x + blockIdx.x * blockDim.x ; //Define id with thread id
	int bid = blockIdx.x;
	int tid = threadIdx.x;
	__shared__ int start;
	__shared__ int end;
	__shared__ int neb[SHARED_MEM];

	//int start = g_row_ptr[bid];
	//int end = g_row_ptr[bid+1]-1;
	//int index = reordered_array[bid];
	if(tid ==0)
	{
		start = g_row_ptr[bid];
		end = g_row_ptr[bid+1]-1;
	}
	__syncthreads();
	int size_list1 = end - start;
	int triangle = 0;
	//if(size_list1 ==0 ) return;
	if(size_list1 < N_THREADS_PER_BLOCK)
	{
		if(tid <= size_list1)
		{
			neb[tid] = g_col_index[tid+start];
		}
		__syncthreads();
		for( int i = 0; i <= size_list1; i++)
		{
			int start2 = g_row_ptr[neb[i]];
			int end2 = g_row_ptr[neb[i]+1]-1;
			int size_list2 = end2 - start2;
			int M = ceil((float)(size_list2 +1)/N_THREADS_PER_BLOCK);
			for( int k = 0; k < M; k++)
			{
				int id = N_THREADS_PER_BLOCK * k + tid;
				int lo = 0;
				int hi = size_list1;
				if(id <= size_list2) 
				{
					int result = 0;
					result = Search(g_col_index[id+start2],neb,size_list1,lo,hi);
					//printf("\nedge(%d , %d) : %d , tid : %d, size_list1 :%d , size_list2: %d, start2 :%d , end2 :%d skey:%d, neb[0]:%d ,neb[%d]:%d",bid, neb[i], result,tid,size_list1+1,size_list2+1,start2,end2,g_col_index[id+start2],neb[0],size_list1,neb[size_list1]);
					//atomicAdd(&g_sum[0],result);
					//printf("\nedge(%d , %d) src : %d dst :%d ", bid,neb[i],size_list1+1,size_list2+1);
					if(result > 0 && result != index)
						triangle = triangle +1;
				}
			}
		}
	}
	else
	{
		int N = ceil((float)(size_list1 +1)/ N_THREADS_PER_BLOCK);
		int remining_size = size_list1;
		int size = N_THREADS_PER_BLOCK-1;
		for( int i = 0; i < N; i++)
		{
			int id = N_THREADS_PER_BLOCK * i + tid;
			int index = 0;
			if( remining_size > size)
			{
				if(id <= size_list1)
				{
					neb[tid] = g_col_index[id+start];
					//printf(" neb : %d", neb[tid]);
				}
				__syncthreads();
				for( int j = start; j <= end; j++)
				{
					int start2 = g_row_ptr[g_col_index[j]];
					int end2 = g_row_ptr[g_col_index[j]+1]-1;
					int size_list2 = end2 - start2;
					int hi = size;
					int lo = index;
					int M = ceil((float)(size_list2 +1)/N_THREADS_PER_BLOCK);
					for( int k = 0; k < M; k++)
					{
						int tempid = N_THREADS_PER_BLOCK * k + tid;
						if(tempid <= size_list2) 
						{
							int result = 0;
							result = Search(g_col_index[tempid+start2],neb,size,lo,hi);
							//printf("\nedge(%d , %d) : %d , tid : %d, size_list1 :%d , size_list2: %d, start2 :%d , end2 :%d, id :%d, skey :%d, N:%d, I:%d, remining_size:%d, size:%d, neb[0]:%d, neb[%d]:%d if ",bid, g_col_index[j], result,tid,size_list1+1,size_list2+1,start2,end2,id,g_col_index[tempid+start2],N,i,remining_size,size,neb[0],size,neb[size]);
							//atomicAdd(&g_sum[0],result);
							//printf("\nedge(%d , %d) src : %d dst :%d ", bid,g_col_index[j],size_list1+1,size_list2+1);
							if(result > 0 && result != index)
								triangle = triangle+1;
						}
					}
				}
				__syncthreads();
				remining_size = remining_size-(size+1);
			}
			else
			{

				if(id <= size_list1)
				{
					neb[tid] = g_col_index[id+start];
					//printf(" neb : %d", neb[tid]);
				}
				__syncthreads();
				for( int j = start; j <= end; j++)
				{
					int start2 = g_row_ptr[g_col_index[j]];
					int end2 = g_row_ptr[g_col_index[j]+1]-1;
					int size_list2 = end2 - start2;
					int M = ceil((float)(size_list2 +1)/ N_THREADS_PER_BLOCK);
					for (int k = 0; k < M; k++)
					{
						int tempid = N_THREADS_PER_BLOCK * k + tid;
						if(tempid <= size_list2) 
						{
							int result = 0;
							result = Search(g_col_index[tempid+start2],neb,remining_size);
							//printf("\nedge(%d , %d) : %d , tid : %d, size_list1 :%d , size_list2: %d, start2 :%d , end2 :%d, id :%d, skey :%d, N:%d, I:%d neb[0]:%d, neb[%d]:%d, else",bid, g_col_index[j], result,tid,size_list1+1,size_list2+1,start2,end2,id,g_col_index[tempid+start2],N,i,neb[0],remining_size,neb[remining_size]);
							//atomicAdd(&g_sum[0],result);
							//printf("\nedge(%d , %d) src : %d dst :%d ", bid,g_col_index[j],size_list1+1,size_list2+1);
							triangle += result;
						}
					}
				}
			}
			__syncthreads();
		}
	}
	atomicAdd(&g_sum[0],triangle);
}
int main(int argc, char *argv[])
{
	hipEvent_t start3,stop3;
	hipEventCreate(&start3);
	hipEventCreate(&stop3);

	int *col_index;   //CPU MEMORY ALLOCATION
	col_index = (int *)malloc(sizeof(int)*NUM_VERTICES);
	int *g_col_index;  //GPU MEMORY ALOOCATION
	hipMalloc(&g_col_index,sizeof(int)*NUM_VERTICES);

	int *row_ptr;  //CPU MEMORY ALLOCATION
	row_ptr = (int *)malloc(sizeof(int)*NUM_EDGES);
	int *g_row_ptr;   // GPU MEMORY ALLOCATION
	hipMalloc(&g_row_ptr,sizeof(int)*NUM_EDGES);

	int edge=0,data=0; //vertex=10670, data allocation from file..

	char *argument2 = argv[2]; //take argument from terminal and initilize
	int vertex=atoi(argument2); //initilize variable

	int *g_sum;
	int *sum;
	sum= (int *)malloc(sizeof(int)*1);
	hipMalloc((void**)&g_sum,sizeof(int)*1);

	//int nblocks = ceil((float)vertex / BLOCKSIZE);

	//**********file operations***************
	FILE *file;
	file = fopen(argv[1],"r");

	//******************Data From File*******************
	if(file == NULL)
	{
		printf("file not opened\n");
		exit(0);
	}
	else
	{
		fscanf(file , "%d", &edge);
		//printf("\nRow_ptr :");
		for(int i=0; i<=vertex+1; i++)
		{
			fscanf(file, "%d", &data);
			row_ptr[i]=data;
			//printf(" %d",data);
		}
		//printf("\nCol_index :");
		for(int j=0; j<edge; j++)
		{
			fscanf(file,"%d", &data);
			col_index[j]=data;
			//printf(" %d",data);
		}
	}
	//**** SEND DATA CPU TO GPU *********************
	hipMemcpy(g_col_index,col_index,sizeof(int)*NUM_VERTICES,hipMemcpyHostToDevice);
	hipMemcpy(g_row_ptr,row_ptr,sizeof(int)*NUM_EDGES,hipMemcpyHostToDevice);

	//****************KERNEL CALLED *****************
	hipEventRecord(start3);
	Find_Triangle<<<vertex,N_THREADS_PER_BLOCK>>>(g_col_index,g_row_ptr,vertex,edge,g_sum);
	hipEventRecord(stop3);
	hipDeviceSynchronize();
	hipMemcpy(sum,g_sum,sizeof(int)*1,hipMemcpyDeviceToHost);
	int Triangle = sum[0];

	hipEventSynchronize(stop3);
	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start3, stop3);
	//printf("\nSearch : %.4f sec ",milliseconds/1000);
	printf("\nSearch : %.6f sec Vertex : %d Edge : %d Triangle : %d ",milliseconds/1000,vertex,edge*2,Triangle);


	//********** FREE THE MEMORY BLOCKS *****************
	free(col_index);
	free(row_ptr);
	hipFree(g_col_index);
	hipFree(g_row_ptr);
	printf("\n");
	return 0;
}
