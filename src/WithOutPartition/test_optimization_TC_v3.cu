#include<stdio.h>
#include<stdlib.h>
#include<hip/hip_runtime.h>
#include<math.h>
#include<time.h>

//#define N_THREADS_PER_BLOCK 256
//#define SHARED_MEM 1024

inline hipError_t checkCuda(hipError_t result)
{
#if defined(DEBUG) || defined(_DEBUG)
  if (result != cudaSuccess) {
    printf(stderr, "CUDA Runtime Error: %s\n", cudaGetErrorString(result));
    assert(result == cudaSuccess);
  }
#endif
  return result;
}

#define N_THREADS_PER_BLOCK 1024
#define SHARED_MEM_SIZE (N_THREADS_PER_BLOCK * sizeof(unsigned long long int))
__device__ unsigned long long int Search(unsigned long long int skey, unsigned long long int *arr, unsigned long long int end, unsigned long long int start, unsigned long long int index2, unsigned long long int *g_row_ptr)
{
    for(unsigned long long int i = start; i <= end; i++)
    {
        if(skey == arr[i])
        {
            unsigned long long int start3 = g_row_ptr[skey];
            unsigned long long int end3 = g_row_ptr[skey+1]-1;
            for(unsigned long long int j = start3; j <= end3; j++)
            {
                if(arr[j] == index2)
                {
                    return 1;
                }
            }
        }
    }

    return 0;
}
__global__ void Find_Triangle(unsigned long long int *g_col_index, unsigned long long int *g_row_ptr, unsigned long long int *g_sum )
{
    unsigned long long int bid = blockIdx.x;
    unsigned long long int tid = threadIdx.x;

    unsigned long long int start = g_row_ptr[bid];
    unsigned long long int end = g_row_ptr[bid+1]-1;
    unsigned long long int size_list1 = end - start;

    extern __shared__ unsigned long long int sdata[];
    unsigned long long int *shared_col_index = (unsigned long long int *)sdata;
    for(unsigned long long int i = tid; i < size_list1; i += N_THREADS_PER_BLOCK)
    {
        shared_col_index[i] = g_col_index[start + i];
    }
    __syncthreads();

    unsigned long long int triangle = 0;
    for(unsigned long long int i = start; i <= end; i++)
    {
        unsigned long long int start2 = g_row_ptr[g_col_index[i]];
        unsigned long long int end2 = g_row_ptr[g_col_index[i]+1]-1;
        unsigned long long int size_list2 = end2 - start2;

        unsigned long long int M = ceil((float)(size_list2+1)/N_THREADS_PER_BLOCK);
        for(unsigned long long int k = 0; k < M; k++)
        {
            unsigned long long int id = N_THREADS_PER_BLOCK * k + tid;
            unsigned long long int index2 = id + start2;

            if(id <= size_list2)
            {
                unsigned long long int result = 0;
                for(unsigned long long int j = 0; j < size_list1; j++)
                {
                    unsigned long long int skey = shared_col_index[j];
                    result += Search(skey, g_col_index, end, start, index2, g_row_ptr);
                }
                triangle += result;
            }
        }
    }

    atomicAdd(&g_sum[0],triangle);
}


int main(int argc, char *argv[])
{
	hipEvent_t start3,stop3;
	hipEventCreate(&start3);
	hipEventCreate(&stop3);

	unsigned long long int Edges=0,data=0,Vertex=0, row_ptr_s=0, col_idx_s=0; //vertex=10670, data allocation from file..
	//**********file operations***************
	FILE *file;
	file = fopen(argv[1],"r");
	//******************Data From File*******************
	if(file == NULL)
	{
		printf("file not opened\n");
		exit(0);
	}
	else
	{
    fscanf(file , "%llu", &Vertex);
    fscanf(file , "%llu", &Edges);
		fscanf(file , "%llu", &row_ptr_s);
		fscanf(file , "%llu", &col_idx_s);

		unsigned long long int *row_ptr;  //CPU MEMORY ALLOCATION
		row_ptr = (unsigned long long int *)malloc(sizeof(unsigned long long int)*row_ptr_s);
    unsigned long long int *col_index;   //CPU MEMORY ALLOCATION
    col_index = (unsigned long long int *)malloc(sizeof(unsigned long long int)*col_idx_s);

		for(unsigned long long int i=0; i<row_ptr_s; i++)
		{
			fscanf(file, "%llu", &data);
			row_ptr[i]=data;
			//printf(" %llu",data);
		}
		//printf("\nCol_index :");
		for(unsigned long long int j=0; j<col_idx_s; j++)
		{
			fscanf(file,"%llu", &data);
			col_index[j]=data;
			//printf(" %llu",data);
		}

		unsigned long long int *g_row_ptr;   // GPU MEMORY ALLOCATION
		hipMalloc(&g_row_ptr,sizeof(unsigned long long int)*row_ptr_s);
    unsigned long long int *g_col_index;  //GPU MEMORY ALOOCATION
		hipMalloc(&g_col_index,sizeof(unsigned long long int)*col_idx_s);
    // int *neb;  //GPU MEMORY ALOOCATION
    // cudaMalloc(&neb,sizeof( int)*N_THREADS_PER_BLOCK);

		//**** SEND DATA CPU TO GPU *********************
    hipMemcpy(g_row_ptr,row_ptr,sizeof(unsigned long long int)*row_ptr_s,hipMemcpyHostToDevice);
		hipMemcpy(g_col_index,col_index,sizeof(unsigned long long int)*col_idx_s,hipMemcpyHostToDevice);

		unsigned long long int *sum;
		sum = (unsigned long long int *)malloc(sizeof(unsigned long long int)*1);

		unsigned long long int *g_sum;
		hipMalloc((void**)&g_sum,sizeof(unsigned long long int)*1);

		//****************KERNEL CALLED *****************
		hipEventRecord(start3);
		Find_Triangle<<<Vertex,N_THREADS_PER_BLOCK>>>(g_col_index,g_row_ptr,g_sum);
		hipEventRecord(stop3);
		hipDeviceSynchronize();
		hipMemcpy(sum,g_sum,sizeof(unsigned long long int)*1,hipMemcpyDeviceToHost);
		unsigned long long int Triangle = sum[0];

		hipEventSynchronize(stop3);
		float milliseconds = 0;
		hipEventElapsedTime(&milliseconds, start3, stop3);
		//printf("\nSearch : %.4f sec ",milliseconds/1000);
		printf("\nSearch : %.6f sec Vertex : %llu Edge : %llu Triangle : %llu\n",milliseconds/1000,Vertex,col_idx_s,Triangle);

		//********** FREE THE MEMORY BLOCKS *****************
		free(col_index);
		free(row_ptr);
		free(sum);
		hipFree(g_col_index);
		hipFree(g_row_ptr);
		hipFree(g_sum);
	}
	//printf("\n");
	return 0;
}
