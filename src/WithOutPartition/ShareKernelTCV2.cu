#include<stdio.h>
#include<stdlib.h>
#include<hip/hip_runtime.h>
#include<math.h>
#include<time.h>

#define NUM_VERTICES 999999999
#define NUM_EDGES 999999999
#define N_THREADS_PER_BLOCK 64

//-------------------intersection function ----------------------------------
__device__ int Intersection(int src, int dst, int *g_col_index , int *g_row_ptr ,int *neb, int size_list)
{
	int total = 0 ;
	int list1_start = 0;
	int list1_end = size_list;
	int list2_start = g_row_ptr[dst];
	int list2_end = g_row_ptr[dst+1];

	while (list1_start <= list1_end && list2_start < list2_end)
	{
		if (neb[list1_start] < g_col_index[list2_start]) list1_start++ ;
		else if (g_col_index[list2_start] < neb[list1_start]) list2_start++ ;
		else if (neb[list1_start] == g_col_index[list2_start])
		{
			total++;
			list1_start++;
			list2_start++;
		}
	}
	return total; //return total triangles found by each thread...
}

__global__ void Find_Triangle(int *g_col_index, int *g_row_ptr, int vertex, int edge ,int *g_sum )
{
	//int id = threadIdx.x + blockIdx.x * blockDim.x ; //Define id with thread id
	int bid = blockIdx.x;
	int tid = threadIdx.x;
	int start = g_row_ptr[bid];
	int end = g_row_ptr[bid+1]-1;
	int size_list = end - start;

	if(size_list < N_THREADS_PER_BLOCK)
	{
		__shared__ int neb[N_THREADS_PER_BLOCK];
		if(tid <= size_list)
		{
			neb[tid] = g_col_index[tid+start];
		}
		__syncthreads();
		if(tid <= size_list)
		{
			int total = 0;
			total = Intersection(bid, neb[tid], g_col_index, g_row_ptr,neb,size_list);
			//printf("\nedge(%d , %d) : %d",bid, neb[tid], total);
			atomicAdd(&g_sum[0],total);
		}
	}
	//else if(size_list > N_THREADS_PER_BLOCK && size_list <= N_THREADS_PER_BLOCK * 8)
	else
	{
		__shared__ int neb[2048];
		int N = ceil((float)size_list / N_THREADS_PER_BLOCK);
		printf("size_list :%d , N :%d\n",size_list,N);
		for(int i = 0; i < N ; i++)
		{
			int id = N_THREADS_PER_BLOCK * i + tid;
			if( id <= size_list)
			{
				neb[id] = g_col_index[id + start];
			}
		}
		__syncthreads();
		for (int i = 0; i < N; i++)
		{
			int id = N_THREADS_PER_BLOCK * i + tid;
			if( id <= size_list)
			{
				int total = 0;
				total = Intersection(bid, neb[tid], g_col_index, g_row_ptr,neb,size_list);
				//printf("\nedge(%d , %d) : %d",bid, neb[tid], total);
				atomicAdd(&g_sum[0],total);
			}
		}
		__syncthreads();
	}/*
	else if (size_list > N_THREADS_PER_BLOCK *8 && size_list <= N_THREADS_PER_BLOCK * 16)
	{
		__shared__ int neb[1024];
		int N = ceil((float)size_list / N_THREADS_PER_BLOCK);
		//printf("size_list :%d , N :%d\n",size_list,N);
		for(int i = 0; i < N ; i++)
		{
			int id = N_THREADS_PER_BLOCK * i + tid;
			if( id <= size_list)
			{
				neb[id] = g_col_index[id + start];
			}
			__syncthreads();
			if( id <= size_list)
			{
				int total = 0;
				total = Intersection(bid, neb[tid], g_col_index, g_row_ptr,neb,size_list);
				//printf("\nedge(%d , %d) : %d",bid, neb[tid], total);
				atomicAdd(&g_sum[0],total);
			}
		}
	}
	else
	{
		__shared__ int neb[2048];
		int N = ceil((float)size_list / N_THREADS_PER_BLOCK);
		printf("size_list :%d , N :%d\n",size_list,N);
		for(int i = 0; i < N ; i++)
		{
			int id = N_THREADS_PER_BLOCK * i + tid;
			if( id <= size_list)
			{
				neb[id] = g_col_index[id + start];
			}
			__syncthreads();
			if( id <= size_list)
			{
				int total = 0;
				total = Intersection(bid, neb[tid], g_col_index, g_row_ptr,neb,size_list);
				//printf("\nedge(%d , %d) : %d",bid, neb[tid], total);
				atomicAdd(&g_sum[0],total);
			}
		}
	}*/
}
int main(int argc, char *argv[])
{
	hipEvent_t start3,stop3;
	hipEventCreate(&start3);
	hipEventCreate(&stop3);

	int *col_index;   //CPU MEMORY ALLOCATION
	col_index = (int *)malloc(sizeof(int)*NUM_VERTICES);
	int *g_col_index;  //GPU MEMORY ALOOCATION
	hipMalloc(&g_col_index,sizeof(int)*NUM_VERTICES);

	int *row_ptr;  //CPU MEMORY ALLOCATION
	row_ptr = (int *)malloc(sizeof(int)*NUM_EDGES);
	int *g_row_ptr;   // GPU MEMORY ALLOCATION
	hipMalloc(&g_row_ptr,sizeof(int)*NUM_EDGES);

	int edge=0,data=0; //vertex=10670, data allocation from file..

	char *argument2 = argv[2]; //take argument from terminal and initilize
	int vertex=atoi(argument2); //initilize variable

	int *g_sum;
	int *sum;
	sum= (int *)malloc(sizeof(int)*1);
	hipMalloc((void**)&g_sum,sizeof(int)*1);

	//int nblocks = ceil((float)vertex / BLOCKSIZE);

	//**********file operations***************
	FILE *file;
	file = fopen(argv[1],"r");

	//******************Data From File*******************
	if(file == NULL)
	{
		printf("file not opened\n");
		exit(0);
	}
	else
	{
		fscanf(file , "%d", &edge);
		//printf("\nRow_ptr :");
		for(int i=0; i<vertex+1; i++)
		{
			fscanf(file, "%d", &data);
			row_ptr[i]=data;
			//printf(" %d",data);
		}
		//printf("\nCol_index :");
		for(int j=0; j<edge; j++)
		{
			fscanf(file,"%d", &data);
			col_index[j]=data;
			//printf(" %d",data);
		}
	}
	//**** SEND DATA CPU TO GPU *********************
	hipMemcpy(g_col_index,col_index,sizeof(int)*NUM_VERTICES,hipMemcpyHostToDevice);
	hipMemcpy(g_row_ptr,row_ptr,sizeof(int)*NUM_EDGES,hipMemcpyHostToDevice);

	//****************KERNEL CALLED *****************
	hipEventRecord(start3);
	Find_Triangle<<<vertex,N_THREADS_PER_BLOCK>>>(g_col_index,g_row_ptr,vertex,edge,g_sum);
	hipEventRecord(stop3);
	hipDeviceSynchronize();
	hipMemcpy(sum,g_sum,sizeof(int)*1,hipMemcpyDeviceToHost);
	int Triangle = sum[0];

	hipEventSynchronize(stop3);
	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start3, stop3);
	printf("\nSearch : %.4f sec ",milliseconds/1000);
	printf("\tVertex : %d\tEdge : %d\tTriangle : %d ",vertex,edge*2,Triangle);


	//********** FREE THE MEMORY BLOCKS *****************
	free(col_index);
	free(row_ptr);
	hipFree(g_col_index);
	hipFree(g_row_ptr);
	printf("\n");
	return 0;
}
