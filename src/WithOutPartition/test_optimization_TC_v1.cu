#include<stdio.h>
#include<stdlib.h>
#include<hip/hip_runtime.h>
#include<math.h>
#include<time.h>

#define N_THREADS_PER_BLOCK 256
#define SHARED_MEM 2048

inline hipError_t checkCuda(hipError_t result)
{
#if defined(DEBUG) || defined(_DEBUG)
  if (result != cudaSuccess) {
    printf(stderr, "CUDA Runtime Error: %s\n", cudaGetErrorString(result));
    assert(result == cudaSuccess);
  }
#endif
  return result;
}

//-------------------intersection function ----------------------------------
__device__ int Search (unsigned long long int skey , unsigned long long int *neb, unsigned long long int sizelist)
{
	unsigned long long int total = 0;
	if(skey < neb[0] || skey > neb[sizelist])
	{
		return 0;
	}
	else if(skey == neb[0] || skey == neb[sizelist])
	{
		return 1;
	}
	else
	{
		unsigned long long int lo = 1;
		unsigned long long int hi = sizelist-1;
		unsigned long long int mid=0;
		while( lo <= hi)
		{
			mid = (hi+lo)/2;
			//printf("\nskey :%llu , mid : %llu ",skey,neb[mid]);
			if( neb[mid] < skey){lo=mid+1;}
			else if(neb[mid] > skey){hi=mid-1;}
			else if(neb[mid] == skey)
			{
				total++;
				break;
			}
		}
	}
	return total;
}
__global__ void Find_Triangle(unsigned long long int *g_col_index, unsigned long long int *g_row_ptr ,unsigned long long int *g_sum )
{
	//unsigned long long int id = threadIdx.x + blockIdx.x * blockDim.x ; //Define id with thread id
	unsigned long long int bid = blockIdx.x;
	unsigned long long int tid = threadIdx.x;
	__shared__ unsigned long long int start;
	__shared__ unsigned long long int end;
	__shared__ unsigned long long int neb[SHARED_MEM];

	if(tid ==0)
	{
		start = g_row_ptr[bid];
		end = g_row_ptr[bid+1]-1;
		//printf("hello\n");
	}
	__syncthreads();
	unsigned long long int size_list1 = end - start;
	unsigned long long int triangle = 0;
	//if(size_list1 ==0 ) return;
	// if(size_list1 < SHARED_MEM)
	// {
	// 	if(tid <= size_list1)
	// 	{
	// 		neb[tid] = g_col_index[tid+start];
	// 	}
	// 	__syncthreads();
	// 	for( unsigned long long int i = 0; i <= size_list1; i++)
	// 	{
	// 		unsigned long long int start2 = g_row_ptr[neb[i]];
	// 		unsigned long long int end2 = g_row_ptr[neb[i]+1]-1;
	// 		unsigned long long int size_list2 = end2 - start2;
	// 		unsigned long long int M = ceil((float)(size_list2 +1)/N_THREADS_PER_BLOCK);
	// 		for( unsigned long long int k = 0; k < M; k++)
	// 		{
	// 			unsigned long long int id = N_THREADS_PER_BLOCK * k + tid;
	// 			if(id <= size_list2)
	// 			{
	// 				unsigned long long int result = 0;
	// 				result = Search(g_col_index[id+start2],neb,size_list1);
	// 				//printf("\nedge(%llu , %llu) : %llu , tid : %llu, size_list1 :%llu , size_list2: %llu, start2 :%llu , end2 :%llu skey:%llu, neb[0]:%llu ,neb[%llu]:%llu",bid, neb[i], result,tid,size_list1+1,size_list2+1,start2,end2,g_col_index[id+start2],neb[0],size_list1,neb[size_list1]);
	// 				//atomicAdd(&g_sum[0],result);
	// 				//pritf("\nedge(%llu , %llu) src : %llu dst :%llu ", bid,neb[i],size_list1+1,size_list2+1);
	// 				triangle += result;
	// 			}
	// 		}
	// 	}
	// }
	// else
//	{
		unsigned long long int N = ceil((float)(size_list1 +1)/ SHARED_MEM);
		unsigned long long int remining_size = size_list1;
		unsigned long long int size = SHARED_MEM-1;
		for( unsigned long long int i = 0; i < N; i++)
		{
			unsigned long long int id = SHARED_MEM * i + tid;
			if( remining_size > size)
			{
        for (unsigned long long int k=0; k<= size_list1; k+=N_THREADS_PER_BLOCK)
        {
  				if(id <= size_list1)
  				{
  					neb[id] = g_col_index[id+start];
  					//printf(" neb : %llu", neb[tid]);
  				}
        }
				__syncthreads();
				for( unsigned long long int j = start; j <= end; j++)
				{
					unsigned long long int start2 = g_row_ptr[g_col_index[j]];
					unsigned long long int end2 = g_row_ptr[g_col_index[j]+1]-1;
					unsigned long long int size_list2 = end2 - start2;
					unsigned long long int M = ceil((float)(size_list2 +1)/N_THREADS_PER_BLOCK);
					for( unsigned long long int k = 0; k < M; k++)
					{
						unsigned long long int tempid = N_THREADS_PER_BLOCK * k + tid;
						if(tempid <= size_list2)
						{
							unsigned long long int result = 0;
							result = Search(g_col_index[tempid+start2],neb,size);
							//printf("\nedge(%llu , %llu) : %llu , tid : %llu, size_list1 :%llu , size_list2: %llu, start2 :%llu , end2 :%llu, id :%llu, skey :%llu, N:%llu, I:%llu, remining_size:%llu, size:%llu, neb[0]:%llu, neb[%llu]:%llu if ",bid, g_col_index[j], result,tid,size_list1+1,size_list2+1,start2,end2,id,g_col_index[tempid+start2],N,i,remining_size,size,neb[0],size,neb[size]);
							//atomicAdd(&g_sum[0],result);
							//printf("\nedge(%llu , %llu) src : %llu dst :%llu ", bid,g_col_index[j],size_list1+1,size_list2+1);
							triangle += result;
						}
					}
				}
				__syncthreads();
				remining_size = remining_size-(size+1);
			}
			else
			{

				if(id <= size_list1)
				{
					neb[tid] = g_col_index[id+start];
					//printf(" neb : %llu", neb[tid]);
				}
				__syncthreads();
				for( unsigned long long int j = start; j <= end; j++)
				{
					unsigned long long int start2 = g_row_ptr[g_col_index[j]];
					unsigned long long int end2 = g_row_ptr[g_col_index[j]+1]-1;
					unsigned long long int size_list2 = end2 - start2;
					unsigned long long int M = ceil((float)(size_list2 +1)/ N_THREADS_PER_BLOCK);
					for (unsigned long long int k = 0; k < M; k++)
					{
						unsigned long long int tempid = N_THREADS_PER_BLOCK * k + tid;
						if(tempid <= size_list2)
						{
							unsigned long long int result = 0;
							result = Search(g_col_index[tempid+start2],neb,remining_size);
							//printf("\nedge(%llu , %llu) : %llu , tid : %llu, size_list1 :%llu , size_list2: %llu, start2 :%llu , end2 :%llu, id :%llu, skey :%llu, N:%llu, I:%llu neb[0]:%llu, neb[%llu]:%llu, else",bid, g_col_index[j], result,tid,size_list1+1,size_list2+1,start2,end2,id,g_col_index[tempid+start2],N,i,neb[0],remining_size,neb[remining_size]);
							//atomicAdd(&g_sum[0],result);
							//printf("\nedge(%llu , %llu) src : %llu dst :%llu ", bid,g_col_index[j],size_list1+1,size_list2+1);
							triangle += result;
						}
					}
				}
			}
			__syncthreads();
		}
//	}
	atomicAdd(&g_sum[0],triangle);
}
int main(int argc, char *argv[])
{
	hipEvent_t start3,stop3;
	hipEventCreate(&start3);
	hipEventCreate(&stop3);

	unsigned long long int Edges=0,data=0,Vertex=0, row_ptr_s=0, col_idx_s=0; //vertex=10670, data allocation from file..

	// char *argument2 = argv[2]; //take argument from terminal and initilize
	// vertex=atoi(argument2);

	//**********file operations***************
	FILE *file;
	file = fopen(argv[1],"r");

	//******************Data From File*******************
	if(file == NULL)
	{
		printf("file not opened\n");
		exit(0);
	}
	else
	{
    fscanf(file , "%llu", &Vertex);
    fscanf(file , "%llu", &Edges);
		fscanf(file , "%llu", &row_ptr_s);
		fscanf(file , "%llu", &col_idx_s);

		unsigned long long int *row_ptr;  //CPU MEMORY ALLOCATION
		row_ptr = (unsigned long long int *)malloc(sizeof(unsigned long long int)*row_ptr_s);
    unsigned long long int *col_index;   //CPU MEMORY ALLOCATION
    col_index = (unsigned long long int *)malloc(sizeof(unsigned long long int)*col_idx_s);

		//printf("\nRow_ptr :");
		for(unsigned long long int i=0; i<row_ptr_s; i++)
		{
			fscanf(file, "%llu", &data);
			row_ptr[i]=data;
			//printf(" %llu",data);
		}
		//printf("\nCol_index :");
		for(unsigned long long int j=0; j<col_idx_s; j++)
		{
			fscanf(file,"%llu", &data);
			col_index[j]=data;
			//printf(" %llu",data);
		}

		unsigned long long int *g_row_ptr;   // GPU MEMORY ALLOCATION
		hipMalloc(&g_row_ptr,sizeof(unsigned long long int)*row_ptr_s);
    unsigned long long int *g_col_index;  //GPU MEMORY ALOOCATION
		hipMalloc(&g_col_index,sizeof(unsigned long long int)*col_idx_s);

		//**** SEND DATA CPU TO GPU *********************
    hipMemcpy(g_row_ptr,row_ptr,sizeof(unsigned long long int)*row_ptr_s,hipMemcpyHostToDevice);
		hipMemcpy(g_col_index,col_index,sizeof(unsigned long long int)*col_idx_s,hipMemcpyHostToDevice);

		unsigned long long int *sum;
		sum = (unsigned long long int *)malloc(sizeof(unsigned long long int)*1);

		unsigned long long int *g_sum;
		hipMalloc((void**)&g_sum,sizeof(unsigned long long int)*1);

		//****************KERNEL CALLED *****************
		hipEventRecord(start3);
		Find_Triangle<<<Vertex,N_THREADS_PER_BLOCK>>>(g_col_index,g_row_ptr,g_sum);
		hipEventRecord(stop3);
		hipDeviceSynchronize();
		hipMemcpy(sum,g_sum,sizeof(unsigned long long int)*1,hipMemcpyDeviceToHost);
		unsigned long long int Triangle = sum[0];

		hipEventSynchronize(stop3);
		float milliseconds = 0;
		hipEventElapsedTime(&milliseconds, start3, stop3);
		//printf("\nSearch : %.4f sec ",milliseconds/1000);
		printf("\nSearch : %.6f sec Vertex : %llu Edge : %llu Triangle : %llu\n",milliseconds/1000,Vertex,col_idx_s,Triangle);


		//********** FREE THE MEMORY BLOCKS *****************
		free(col_index);
		free(row_ptr);
		free(sum);
		hipFree(g_col_index);
		hipFree(g_row_ptr);
		hipFree(g_sum);
	}
	//printf("\n");
	return 0;
}
