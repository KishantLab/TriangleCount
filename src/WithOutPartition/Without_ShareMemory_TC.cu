#include<stdio.h>
#include<stdlib.h>
#include<hip/hip_runtime.h>
#include<math.h>
#include<time.h>

#define N_THREADS_PER_BLOCK 256
//#define SHARED_MEM 1024

inline hipError_t checkCuda(hipError_t result)
{
#if defined(DEBUG) || defined(_DEBUG)
  if (result != cudaSuccess) {
    printf(stderr, "CUDA Runtime Error: %s\n", cudaGetErrorString(result));
    assert(result == cudaSuccess);
  }
#endif
  return result;
}

//-------------------intersection function ----------------------------------
__device__ int Search (unsigned long long int skey , unsigned long long int *neb, unsigned long long int end, unsigned long long int start)
{
	unsigned long long int total = 0;
	if(skey < neb[start] || skey > neb[end])
	{
		return 0;
	}
	else if(skey == neb[start] || skey == neb[end])
	{
		return 1;
	}
	else
	{
		unsigned long long int lo = start+1;
		unsigned long long int hi = end-1;
		unsigned long long int mid=0;
		while( lo <= hi)
		{
			mid = (hi+lo)/2;
			//printf("\nskey :%llu , mid : %llu ",skey,neb[mid]);
			if( neb[mid] < skey){lo=mid+1;}
			else if(neb[mid] > skey){hi=mid-1;}
			else if(neb[mid] == skey)
			{
				total++;
				break;
			}
		}
	}
	return total;
}
__global__ void Find_Triangle(unsigned long long int *g_col_index, unsigned long long int *g_row_ptr ,unsigned long long int *g_sum )
{
	//unsigned long long int id = threadIdx.x + blockIdx.x * blockDim.x ; //Define id with thread id
	unsigned long long int bid = blockIdx.x;
	unsigned long long int tid = threadIdx.x;
  unsigned long long int start;
	unsigned long long int end;
	//unsigned long long int neb[N_THREADS_PER_BLOCK];

	//if(tid ==0)
	//{
		start = g_row_ptr[bid];
		end = g_row_ptr[bid+1]-1;
		//printf("hello\n");
	//}
	//__syncthreads();
	unsigned long long int size_list1 = end - start;
	unsigned long long int triangle = 0;
  if(tid ==0)
  //printf("Kernel called\n");
	//if(size_list1 ==0 ) return;
	//if(size_list1 < N_THREADS_PER_BLOCK)
	//{
		// if(tid <= size_list1)
		// {
		// 	neb[tid] = g_col_index[tid+start];
    //   printf("%llu ",neb[tid]);
		// }
		//__syncthreads();
		for( unsigned long long int i = start; i <= end; i++)
		{
			unsigned long long int start2 = g_row_ptr[g_col_index[i]];
			unsigned long long int end2 = g_row_ptr[g_col_index[i]+1]-1;
			unsigned long long int size_list2 = end2 - start2;
			unsigned long long int M = ceil((float)(size_list2+1)/N_THREADS_PER_BLOCK);
			for( unsigned long long int k = 0; k < M; k++)
			{
				unsigned long long int id = N_THREADS_PER_BLOCK * k + tid;
				if(id <= size_list2)
				{
          //if(id==0){printf("insideloop\n");}
					unsigned long long int result = 0;
					result = Search(g_col_index[id+start2],g_col_index,end,start);
          if(id==0)
					printf("\nedge(%llu , %llu) : %llu , tid : %llu, size_list1 :%llu , size_list2: %llu, start2 :%llu , end2 :%llu skey:%llu, g_col_index[0]:%llu ,g_col_index[%llu]:%llu",bid, g_col_index[i], result,tid,size_list1+1,size_list2+1,start2,end2,g_col_index[id+start2],g_col_index[start],size_list1,g_col_index[end]);
					//atomicAdd(&g_sum[0],result);
					//printf("\nedge(%llu , %llu) src : %llu dst :%llu ", bid,neb[i],size_list1+1,size_list2+1);
					triangle += result;
				}
			}
		}
	//}
	// else
	// {
	// 	unsigned long long int N = ceil((float)(size_list1 +1)/ N_THREADS_PER_BLOCK);
	// 	unsigned long long int remining_size = size_list1;
	// 	unsigned long long int size = N_THREADS_PER_BLOCK-1;
	// 	for( unsigned long long int i = 0; i < N; i++)
	// 	{
	// 		unsigned long long int id = N_THREADS_PER_BLOCK * i + tid;
	// 		if( remining_size > size)
	// 		{
	// 			// if(id <= size_list1)
	// 			// {
	// 			// 	neb[tid] = g_col_index[id+start];
	// 			// 	//printf(" neb : %llu", neb[tid]);
	// 			// }
	// 			// __syncthreads();
	// 			for( unsigned long long int j = start; j <= end; j++)
	// 			{
	// 				unsigned long long int start2 = g_row_ptr[g_col_index[j]];
	// 				unsigned long long int end2 = g_row_ptr[g_col_index[j]+1]-1;
	// 				unsigned long long int size_list2 = end2 - start2;
	// 				unsigned long long int M = ceil((float)(size_list2 +1)/N_THREADS_PER_BLOCK);
	// 				for( unsigned long long int k = 0; k < M; k++)
	// 				{
	// 					unsigned long long int tempid = N_THREADS_PER_BLOCK * k + tid;
	// 					if(tempid <= size_list2)
	// 					{
	// 						unsigned long long int result = 0;
	// 						result = Search(g_col_index[tempid+start2],g_col_index,end,start);
	// 						printf("\nedge(%llu , %llu) : %llu , tid : %llu, size_list1 :%llu , size_list2: %llu, start2 :%llu , end2 :%llu, id :%llu, skey :%llu, N:%llu, I:%llu, remining_size:%llu, size:%llu, g_col_index[0]:%llu, g_col_index[%llu]:%llu if ",bid, g_col_index[j], result,tid,size_list1+1,size_list2+1,start2,end2,id,g_col_index[tempid+start2],N,i,remining_size,size,g_col_index[0],size,g_col_index[size]);
	// 						//atomicAdd(&g_sum[0],result);
	// 						//printf("\nedge(%llu , %llu) src : %llu dst :%llu ", bid,g_col_index[j],size_list1+1,size_list2+1);
	// 						triangle += result;
	// 					}
	// 				}
	// 			}
	// 			__syncthreads();
	// 			remining_size = remining_size-(size+1);
	// 		}
	// 		else
	// 		{
  //
	// 			// if(id <= size_list1)
	// 			// {
	// 			// 	neb[tid] = g_col_index[id+start];
	// 			// 	//printf(" neb : %llu", neb[tid]);
	// 			// }
	// 			// __syncthreads();
	// 			for( unsigned long long int j = start; j <= end; j++)
	// 			{
	// 				unsigned long long int start2 = g_row_ptr[g_col_index[j]];
	// 				unsigned long long int end2 = g_row_ptr[g_col_index[j]+1]-1;
	// 				unsigned long long int size_list2 = end2 - start2;
	// 				unsigned long long int M = ceil((float)(size_list2 +1)/ N_THREADS_PER_BLOCK);
	// 				for (unsigned long long int k = 0; k < M; k++)
	// 				{
	// 					unsigned long long int tempid = N_THREADS_PER_BLOCK * k + tid;
	// 					if(tempid <= size_list2)
	// 					{
	// 						unsigned long long int result = 0;
	// 						result = Search(g_col_index[tempid+start2],g_col_index,end,start);
	// 						printf("\nedge(%llu , %llu) : %llu , tid : %llu, size_list1 :%llu , size_list2: %llu, start2 :%llu , end2 :%llu, id :%llu, skey :%llu, N:%llu, I:%llu g_col_index[0]:%llu, g_col_index[%llu]:%llu, else",bid, g_col_index[j], result,tid,size_list1+1,size_list2+1,start2,end2,id,g_col_index[tempid+start2],N,i,g_col_index[0],remining_size,g_col_index[remining_size]);
	// 						//atomicAdd(&g_sum[0],result);
	// 						//printf("\nedge(%llu , %llu) src : %llu dst :%llu ", bid,g_col_index[j],size_list1+1,size_list2+1);
	// 						triangle += result;
	// 					}
	// 				}
	// 			}
	// 		}
	// 		__syncthreads();
	// 	}
	// }
	atomicAdd(&g_sum[0],triangle);
}
int main(int argc, char *argv[])
{
	hipEvent_t start3,stop3;
	hipEventCreate(&start3);
	hipEventCreate(&stop3);

	unsigned long long int Edges=0,data=0,Vertex=0, row_ptr_s=0, col_idx_s=0; //vertex=10670, data allocation from file..

	// char *argument2 = argv[2]; //take argument from terminal and initilize
	// vertex=atoi(argument2);

	//**********file operations***************
	FILE *file;
	file = fopen(argv[1],"r");

	//******************Data From File*******************
	if(file == NULL)
	{
		printf("file not opened\n");
		exit(0);
	}
	else
	{
    fscanf(file , "%llu", &Vertex);
    fscanf(file , "%llu", &Edges);
		fscanf(file , "%llu", &row_ptr_s);
		fscanf(file , "%llu", &col_idx_s);

		unsigned long long int *row_ptr;  //CPU MEMORY ALLOCATION
		row_ptr = (unsigned long long int *)malloc(sizeof(unsigned long long int)*row_ptr_s);
    unsigned long long int *col_index;   //CPU MEMORY ALLOCATION
    col_index = (unsigned long long int *)malloc(sizeof(unsigned long long int)*col_idx_s);

		//printf("\nRow_ptr :");
		for(unsigned long long int i=0; i<row_ptr_s; i++)
		{
			fscanf(file, "%llu", &data);
			row_ptr[i]=data;
			//printf(" %llu",data);
		}
		//printf("\nCol_index :");
		for(unsigned long long int j=0; j<col_idx_s; j++)
		{
			fscanf(file,"%llu", &data);
			col_index[j]=data;
			//printf(" %llu",data);
		}

		unsigned long long int *g_row_ptr;   // GPU MEMORY ALLOCATION
		hipMalloc(&g_row_ptr,sizeof(unsigned long long int)*row_ptr_s);
    unsigned long long int *g_col_index;  //GPU MEMORY ALOOCATION
		hipMalloc(&g_col_index,sizeof(unsigned long long int)*col_idx_s);
    // int *neb;  //GPU MEMORY ALOOCATION
    // cudaMalloc(&neb,sizeof( int)*N_THREADS_PER_BLOCK);

		//**** SEND DATA CPU TO GPU *********************
    hipMemcpy(g_row_ptr,row_ptr,sizeof(unsigned long long int)*row_ptr_s,hipMemcpyHostToDevice);
		hipMemcpy(g_col_index,col_index,sizeof(unsigned long long int)*col_idx_s,hipMemcpyHostToDevice);

		unsigned long long int *sum;
		sum = (unsigned long long int *)malloc(sizeof(unsigned long long int)*1);

		unsigned long long int *g_sum;
		hipMalloc((void**)&g_sum,sizeof(unsigned long long int)*1);

		//****************KERNEL CALLED *****************
		hipEventRecord(start3);
		Find_Triangle<<<Vertex,N_THREADS_PER_BLOCK>>>(g_col_index,g_row_ptr,g_sum);
		hipEventRecord(stop3);
		hipDeviceSynchronize();
		hipMemcpy(sum,g_sum,sizeof(unsigned long long int)*1,hipMemcpyDeviceToHost);
		unsigned long long int Triangle = sum[0];

		hipEventSynchronize(stop3);
		float milliseconds = 0;
		hipEventElapsedTime(&milliseconds, start3, stop3);
		//printf("\nSearch : %.4f sec ",milliseconds/1000);
		printf("\nSearch : %.6f sec Vertex : %llu Edge : %llu Triangle : %llu\n",milliseconds/1000,Vertex,col_idx_s,Triangle);


		//********** FREE THE MEMORY BLOCKS *****************
		free(col_index);
		free(row_ptr);
		free(sum);
		hipFree(g_col_index);
		hipFree(g_row_ptr);
		hipFree(g_sum);
	}
	//printf("\n");
	return 0;
}
