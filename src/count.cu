
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>

#define NUM_VERTICES 9999999999
#define NUM_EDGES 9999999999
#define BLOCKSIZE 1024

__device__ int intersection(int src, int dst, int *d_col_index, int *d_row_ptr, int *d_vertex_arr ,int v_pos,int id ,int total_v_in_partitions, int NP)
{
  //******initilized Variables *****************
  printf("\npart : %d, src : %d , dst : %d ",NP,src,dst);
  int total = 0 ;
  int pointer1_start = d_row_ptr[id];
  int pointer1_end = d_row_ptr[id+1];

  int index = 0;
  int flag = 0;

  int low = id , high = total_v_in_partitions,  mid;
  while(high - low > 1)
  {
    mid = ( high+low )/2;
    if ( d_vertex_arr[mid] < dst ){ low = mid; }
    else if ( d_vertex_arr[mid] > dst ){ high = mid; }
    else
    {
      index = mid;
      flag++;
      break;
    }
  }

  if (flag == 0)
  {
    for(int i=total_v_in_partitions; i<v_pos; i++)
    {
      if( d_vertex_arr[i] == dst){index = i ;}
    }
  }

  int pointer2_start = d_row_ptr[index];
  int pointer2_end = d_row_ptr[index+1];

  while (pointer1_start < pointer1_end && pointer2_start < pointer2_end)
  {
    if (d_col_index[pointer1_start] < d_col_index[pointer2_start]) pointer1_start++ ;
    else if (d_col_index[pointer2_start] < d_col_index [pointer1_start]) pointer2_start++ ;
    else if (d_col_index[pointer1_start] == d_col_index[pointer2_start])
    {
      total++;
      pointer1_start++;
      pointer2_start++;
    }
  }

  return total; //return total triangles found by each thread...
}

__global__ void Find_Triangle(int *d_col_index, int *d_row_ptr, int *d_vertex_arr,int total_v_in_partitions, int v_pos, int rp_pos, int ci_pos, int *d_sum, int NP )
{
  int id = threadIdx.x + blockIdx.x * blockDim.x ; //Define id with thread id

  if (id < total_v_in_partitions) // only number of vertex thread executed ...
  {
    for (int i = d_row_ptr[id] ; i < d_row_ptr[id+1] ; i++)
    {
      int total = 0;

      //******CALLED INTERSECTION FUNCTION ************
      total = intersection(d_vertex_arr[id], d_col_index[i], d_col_index, d_row_ptr, d_vertex_arr ,v_pos ,id ,total_v_in_partitions,NP);
      atomicAdd(&d_sum[0],total);

    }

  //printf("Total Triangle p: %d",d_sum[0]);
  }

}

int main(int argc, char *argv[])
{
  //------initilization of variables------------//
  int Total_Triangle = 0;

  float total_kernel_time = 0.0 ;
  float total_time = 0.0;

  int v_pos, rp_pos, ci_pos, total_v_in_partitions;

  char *argument2 = argv[2]; //take argument from terminal and initilize
  int vertex=atoi(argument2);

  char *argument3 = argv[3]; //take argument from terminal and initilize
  int edge=atoi(argument3);

  char *argument4 = argv[4];
  int no_partitions = atoi(argument4);

  int *new_col_index;
  new_col_index= (int *) malloc(sizeof(int)*edge);

  int *new_row_ptr;
  new_row_ptr = (int *) malloc(sizeof(int)*vertex);

  int *vertex_arr;
  vertex_arr = (int *) malloc(sizeof(int)*vertex);

  int *v_pos_arr;
  v_pos_arr = (int *) malloc(sizeof(int)*vertex);

  int *rp_pos_arr;
  hipHostMalloc(&rp_pos_arr,sizeof(int)*no_partitions+1, hipHostMallocDefault);

  int *ci_pos_arr;
  hipHostMalloc(&ci_pos_arr,sizeof(int)*no_partitions+1, hipHostMallocDefault);

  int *total_v_in_partitions_arr;
  total_v_in_partitions_arr = (int *) malloc(sizeof(int)*no_partitions+1);

  //--------------------Load DATA In Memory---------------------//

  FILE *file;
  file = fopen(argv[1],"r");

  if (file == NULL)
  {
    printf("\nFile Not Operned.........");
    exit(0);
  }
  else
  {
    v_pos_arr[0] = 0;
    ci_pos_arr[0] = 0;
    rp_pos_arr[0] = 0;
    total_v_in_partitions_arr[0] = 0;

    for (int i = 0; i < no_partitions; i++)
    {
      int data = 0;
      fscanf(file, "%d" , &v_pos);
      v_pos_arr[i+1] = v_pos;
      printf("v_pos_arr[%d]:%d  ",i+1,v_pos);

      fscanf(file, "%d" , &rp_pos);
      rp_pos_arr[i+1] = rp_pos;
      printf("rp_pos_arr[%d]: %d  ",i+1,rp_pos);

      fscanf(file, "%d" , &ci_pos);
      ci_pos_arr[i+1] = ci_pos;
      printf("ci_pos_arr[%d]: %d  ",i+1,ci_pos);

      fscanf(file, "%d" , &total_v_in_partitions);
      total_v_in_partitions_arr[i+1] = total_v_in_partitions;
      printf("total_v_in_partitions_arr[%d]: %d  ",i+1,total_v_in_partitions);

      for (int j = v_pos_arr[i] ; j < v_pos_arr[i+1]; j++)
      {
        fscanf(file, "%d", &data);
        vertex_arr[j]=data;
      }
      printf("\nVertex Array Loaded....");
      for (int j = rp_pos_arr[i] ; j < rp_pos_arr[i+1]; j++)
      {
        fscanf(file, "%d", &data);
        new_row_ptr[j]=data;
      }
      printf("\nRow Pointer Array Loaded.....");

      for (int j = ci_pos_arr[i] ; j< ci_pos_arr[i+1]; j++)
      {
        fscanf(file, "%d", &data);
        new_col_index[j]=data;
      }
      printf("\nCol Index Array Loaded.......");
    }
    printf("\n");
  }


//==============================CREATE STREAMS================================//
  hipStream_t stream[no_partitions];
  printf("\nGPU Stream Created.............");

  int *d_col_index;  //GPU MEMORY ALOOCATION
  hipMalloc(&d_col_index,sizeof(int)*ci_pos);

  int *d_vertex_arr;  //GPU MEMORY ALOOCATION
  hipMalloc(&d_vertex_arr,sizeof(int)*v_pos);

  int *d_row_ptr;   // GPU MEMORY ALLOCATION
  hipMalloc(&d_row_ptr,sizeof(int)*rp_pos);

  printf("\nGPU Arrays Created...............");
  int *d_sum;
  int *sum;
  hipHostMalloc(&sum,sizeof(int)*1, hipHostMallocDefault);
  hipMalloc((void**)&d_sum,sizeof(int)*1);

  int nblocks = ceil((float)total_v_in_partitions / BLOCKSIZE);

  printf("\nBlocks : %d ", nblocks);
  hipEvent_t start, stop, startG, stopG;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  printf("\nEvent Initilize......");
  hipEventCreate(&startG);
  printf("\nFirst Event Created.......");
  hipEventCreate(&stopG);
  printf("\nSecend Event created..........");

  for (int i=0; i<no_partitions; i++){hipStreamCreate(&stream[i]);}  //create stream for Device

  //--------copy data from host to device --------------//
  hipEventRecord(start);
  printf("\nStart Copy Data From Host To Device ......");
  for (int j = 0; j < no_partitions; j++)
  {
    printf("\nCopy start For P:%d.....",j);
    int offsetv = v_pos_arr[j+1];
	printf("\nOffset v_pos : %d", offsetv);
    int offsetr = rp_pos_arr[j+1];
    int offsetc = ci_pos_arr[j+1];

    hipMemcpyAsync(d_row_ptr,new_row_ptr,sizeof(int)*offsetr,hipMemcpyHostToDevice,stream[j]);
    hipMemcpyAsync(d_col_index,new_col_index,sizeof(int)*offsetc,hipMemcpyHostToDevice,stream[j]);
    hipMemcpyAsync(d_vertex_arr,vertex_arr+v_pos_arr[j],sizeof(int)*offsetv,hipMemcpyHostToDevice,stream[j]);
    //cudaStreamSynchronize(stream[j]);

    printf("\nCopy Completed for P:%d....",j);
  }
  printf("\nKernel Called...................");
  hipEventRecord(startG);
  printf("\nStart Kernel..........");
  for (int k = 0 ; k < no_partitions; k++)
  {
    //cudaStreamSynchronize(stream[k]);
    printf("\nKernel Called for P:%d....",k);
    Find_Triangle<<<nblocks,BLOCKSIZE,0,stream[k]>>>(d_col_index,d_row_ptr,d_vertex_arr,total_v_in_partitions_arr[k+1],v_pos_arr[k+1],rp_pos_arr[k+1],ci_pos_arr[k+1],d_sum,k);
  }
  hipEventRecord(stopG);
  printf("\nKernel Execution Done .....");

  for( int j=0 ; j < no_partitions; j++)
  {
    hipMemcpyAsync(sum,d_sum,sizeof(int)*1,hipMemcpyDeviceToHost,stream[j]);
   	//int Triangle = sum[0];
   	//printf("\t%d\n" , Triangle);
  	//total_Triangle = Total_Triangle + Triangle ;
  }

  for( int j=0 ; j < no_partitions; j++)
  {
	//cuStreamSynchonize(stream[j]);
	//cudaError_t cudaStreamSynchronize();
	int Triangle = sum[0];

   	//printf("\t%d\n" , Triangle);
  	Total_Triangle = Total_Triangle + Triangle ;

  }
  hipEventRecord(stop);
  hipEventSynchronize(stop);
  float millisecondsG = 0;
  hipEventElapsedTime(&millisecondsG, startG, stopG);
  printf("\nGPU Time : %.4f sec",millisecondsG/1000);
  total_kernel_time = total_kernel_time + millisecondsG/1000;

  float milliseconds = 0;
  hipEventElapsedTime(&milliseconds, start, stop);
  printf("\nTotal Time :%.4f sec",milliseconds/1000);
  total_time = total_time + milliseconds/1000;

  printf("\nTotal Triangle : %d ",Total_Triangle);
  printf("\n");

  return 0;
}
