#include<stdio.h>
#include<stdlib.h>
#include<hip/hip_runtime.h>
#include<math.h>
#include<time.h>

#define NUM_VERTICES 99999999
#define NUM_EDGES 99999999
#define BLOCKSIZE 1024

//-------------------intersection function ----------------------------------
__device__ int JumpIntersection(int src, int dst, int *g_col_index , int *g_row_ptr )
{
	int total = 0 ;
	int list1_start = g_row_ptr[src];
	int list1_end = g_row_ptr[src+1];
	int list2_start = g_row_ptr[dst];
	int list2_end = g_row_ptr[dst+1];
	int sizelist2 = list2_end - list2_start;
	int step = (int)floor(sqrtf(sizelist2));
	for(int x =  list1_start; x < list1_end; x++)
	{
		if(g_col_index[list2_start] > g_col_index[x])
		{
			continue;
		}
		else
		{
			int start = list2_start;
			int end = step + list2_start;
			while( g_col_index[end-1] < g_col_index[x] && start < list2_end)
			{
				start = end;
				end = end +step;
				if( end > list2_end)
				{
					end = list2_end;
					break;
				}
			}
			
			int lo = start , hi = end-1, mi;
			while(lo <= hi)
			{
				mi = (hi+lo)/2;
				if(g_col_index[mi] < g_col_index[x]){lo=mi+1;}
				else if(g_col_index[mi] > g_col_index[x]){hi = mi-1;}	
				else{
					total++;
					break;
				}
			}
			/*

			for( int y = start; y < end; y++)
			{
				if(g_col_index[y] == g_col_index[x])
				{
					total++;
				}
			}
			*/
			
		}
	}
	return total; //return total triangles found by each thread...
}

__global__ void Find_Triangle_Jump(int *g_col_index, int *g_row_ptr, int vertex, int edge ,int *g_sum )
{
	int id = threadIdx.x + blockIdx.x * blockDim.x ; //Define id with thread id
	if (id < vertex) // only number of vertex thread executed ...
	{
		for (int i = g_row_ptr[id] ; i < g_row_ptr[id+1] ; i++)
		{
			int total = 0;
			total = JumpIntersection(id, g_col_index[i], g_col_index, g_row_ptr );
			//printf("\nedge(%d , %d) : %d",id, g_col_index[i], total);
			atomicAdd(&g_sum[0],total);
		}
	}
}

int main(int argc, char *argv[])
{
	hipEvent_t start3,stop3;
	hipEventCreate(&start3);
	hipEventCreate(&stop3);

	int *col_index;   //CPU MEMORY ALLOCATION
	col_index = (int *)malloc(sizeof(int)*NUM_VERTICES);
	int *g_col_index;  //GPU MEMORY ALOOCATION
	hipMalloc(&g_col_index,sizeof(int)*NUM_VERTICES);

	int *row_ptr;  //CPU MEMORY ALLOCATION
	row_ptr = (int *)malloc(sizeof(int)*NUM_EDGES);
	int *g_row_ptr;   // GPU MEMORY ALLOCATION
	hipMalloc(&g_row_ptr,sizeof(int)*NUM_EDGES);

	int edge=0,data=0; //vertex=10670, data allocation from file..

	char *argument2 = argv[2]; //take argument from terminal and initilize
	int vertex=atoi(argument2); //initilize variable

	int *g_sum;
	int *sum;
	sum= (int *)malloc(sizeof(int)*1);
	hipMalloc((void**)&g_sum,sizeof(int)*1);

	int nblocks = ceil((float)vertex / BLOCKSIZE);

	//**********file operations***************
	FILE *file;
	file = fopen(argv[1],"r");

	//******************Data From File*******************
	if(file == NULL)
	{
		printf("file not opened\n");
		exit(0);
	}
	else
	{
		fscanf(file , "%d", &edge);
		//printf("\nRow_ptr :");
		for(int i=0; i<vertex+1; i++)
		{
			fscanf(file, "%d", &data);
			row_ptr[i]=data;
			//printf(" %d",data);
		}
		//printf("\nCol_index :");
		for(int j=0; j<edge; j++)
		{
			fscanf(file,"%d", &data);
			col_index[j]=data;
			//printf(" %d",data);
		}
	}
	//**** SEND DATA CPU TO GPU *********************
	hipMemcpy(g_col_index,col_index,sizeof(int)*NUM_VERTICES,hipMemcpyHostToDevice);
	hipMemcpy(g_row_ptr,row_ptr,sizeof(int)*NUM_EDGES,hipMemcpyHostToDevice);

	//****************KERNEL CALLED *****************
	hipEventRecord(start3);
	Find_Triangle_Jump<<<nblocks,BLOCKSIZE>>>(g_col_index,g_row_ptr,vertex,edge,g_sum);
	hipEventRecord(stop3);
	hipDeviceSynchronize();
	hipMemcpy(sum,g_sum,sizeof(int)*1,hipMemcpyDeviceToHost);
	int Triangle_Jump = sum[0];
	
	hipEventSynchronize(stop3);
	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start3, stop3);
	printf("\nSearch : %.4f sec",milliseconds/1000);
	printf("\tVertex : %d\tEdge : %d\tTriangle : %d ",vertex,edge,Triangle_Jump);


	//********** FREE THE MEMORY BLOCKS *****************
	//free(col_index);
	//free(row_ptr);
	//cudaFree(g_col_index);
	//cudaFree(g_row_ptr);
	printf("\n");
	return 0;
}
